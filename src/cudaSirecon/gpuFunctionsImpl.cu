#include "hip/hip_runtime.h"
#include "gpuFunctionsImpl_hh.cu"

__host__ void image_arithmetic(GPUBuffer* a, const GPUBuffer& b, int offset,
    int len, float alpha, float beta)
{
  float* aPtr = (float*)a->getPtr();
  aPtr += offset;
  const float* bPtr = (const float*)b.getPtr();

  int blockSize = 128;
  int numBlocks = (int)(ceil((float)len / blockSize));
  image_arithmetic_kernel<<<numBlocks, blockSize>>>(aPtr, bPtr,
      len, alpha, beta);
}

__host__ void image_arithmetic(GPUBuffer* a, const GPUBuffer& b,
    int offsetA, int offsetB, int len, float alpha, float beta)
{
  float* aPtr = (float*)a->getPtr();
  aPtr += offsetA;
  const float* bPtr = (const float*)b.getPtr();
  bPtr += offsetB;

  int blockSize = 128;
  int numBlocks = (int)(ceil((float)len / blockSize));
  image_arithmetic_kernel<<<numBlocks, blockSize>>>(aPtr, bPtr,
      len, alpha, beta);
}

__global__ void image_arithmetic_kernel(float* a, const float* b,
    int len, float alpha, float beta)
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < len) {
    a[tid] = alpha * a[tid] + beta * b[tid];
  }
}

__host__ void apodize(int napodize, int nx,int ny, GPUBuffer* image,
    int offset)
{
  int blockSize = 64;
  int numBlocks = (int)(ceil((float)nx / blockSize));
  apodize_x_kernel<<<numBlocks, blockSize>>>(napodize, nx, ny,
      ((float*)image->getPtr()) + offset);

  numBlocks = (int)(ceil((float)ny / blockSize));
  apodize_y_kernel<<<numBlocks, blockSize>>>(napodize, nx, ny,
      ((float*)image->getPtr()) + offset);
}

__global__ void apodize_x_kernel(int napodize, int nx, int ny,
    float* image)
{
  // This blends the top and bottom edges for column k
  int k = blockDim.x * blockIdx.x + threadIdx.x;
  int xdim = (nx/2+1) * 2;
  if (k < nx) {
    float diff = (image[(ny - 1) * xdim + k] - image[k]) / 2.0;
    for (int l = 0; l < napodize; ++l) {
      float fact = 1.0 - sinf((l + 0.5) / (float)napodize * M_PI * 0.5);
      image[l * xdim + k] += diff * fact;
      image[(ny - 1 - l) * xdim + k] -=  diff * fact;
    }
  }
}

__global__ void apodize_y_kernel(int napodize, int nx, int ny,
    float* image)
{
  // This blends the left and right edges for row l
  int l = blockDim.x * blockIdx.x + threadIdx.x;
  int xdim = (nx/2+1) * 2;
  if (l < ny) {
    float diff = (image[l * xdim + nx - 1] - image[l * xdim]) / 2.0;
    for (int k = 0; k < napodize; ++k) {
      float fact = 1.0 - sinf((k + 0.5) / (float)napodize * M_PI * 0.5);
      image[l * xdim + k] += diff * fact;
      image[l * xdim + (nx - 1 - k)] -= diff * fact;
    }
  }
}

__host__ void cosapodize(int nx,int ny, GPUBuffer* image, int offset)
{
  dim3 blockSize;
  blockSize.x = 16;
  blockSize.y = 16;
  blockSize.z = 1;
  dim3 numBlocks;
  numBlocks.x = (int)(ceil((float)nx / blockSize.x));
  numBlocks.y = (int)(ceil((float)ny / blockSize.y));
  numBlocks.z = 1;
  cosapodize_kernel<<<numBlocks, blockSize>>>(nx, ny,
      ((float*)image->getPtr()) + offset, float cosPeriodX, float cosPeriodY, float cosAmpX, float cosAmpY)
}

__global__ void cosapodize_kernel(int nx, int ny, float* image)
{
  int k = blockDim.x * blockIdx.x + threadIdx.x;
  int l = blockDim.y * blockIdx.y + threadIdx.y;
  int xdim = (nx/2+1) * 2;
  if (k<nx && l<ny) {
    float xfact = sinf(M_PI * cosPeriodX * (((float)k + 0.5) / nx)) * cosAmpX;
    float yfact = sinf(M_PI * cosPeriodY * (((float)l + 0.5) / ny)) * cosAmpY;
    image[l * xdim + k] *= xfact * yfact;
  }
}

__host__ void rescale(int nx, int ny, int nz, int z, int zoffset, int direction,
    int wave, int t, int nphases, std::vector<GPUBuffer>* images, int equalizez,
    int equalizet, double* sum_dir0_phase0)
{
  std::vector<float> sum(nphases);

  dim3 blockSize;
  blockSize.x = RED_BLOCK_SIZE_X;
  blockSize.y = RED_BLOCK_SIZE_Y;
  blockSize.z = 1;
  dim3 numBlocks;
  numBlocks.x = (int)(ceil((float)nx / blockSize.x));
  numBlocks.y = (int)(ceil((float)ny / blockSize.y));
  numBlocks.z = 1;
  int xdim = (nx/2 + 1) * 2;

  GPUBuffer sumTmpDev(numBlocks.x * numBlocks.y * sizeof(float), 0);
  CPUBuffer sumTmpHost(numBlocks.x * numBlocks.y * sizeof(float));
  for (int phase = 0; phase < nphases; ++phase) {
    sum_reduction_kernel<<<numBlocks, blockSize>>>(
         ((float*)(images->at(phase).getPtr())) +
         (z + zoffset) * xdim * ny, nx, ny,
        (float*)sumTmpDev.getPtr());
    sumTmpDev.set(&sumTmpHost, 0, sumTmpDev.getSize(), 0);
    sum[phase] = 0.0f;
    for (int i = 0; i < numBlocks.x * numBlocks.y; ++i) {
      sum[phase] += ((float*)sumTmpHost.getPtr())[i];
    }
  }

  if (direction == 0 && !(equalizet && t != 0)) {
    sum_dir0_phase0[wave * nz + z] = (double)sum[0];
  }
  float ref;
  if (equalizez) {
    ref = sum_dir0_phase0[wave * nz + 0];
  } else  {
    ref = sum_dir0_phase0[wave * nz + z];
  }

  for (int phase = 0; phase < nphases; ++phase) {
    float ratio = ref / sum[phase];
    rescale_kernel<<<numBlocks, blockSize>>>(
         (float*)((*images)[phase].getPtr()) +
         (z + zoffset) * xdim * ny, nx, ny,
        ratio);
  }
}

__global__ void sum_reduction_kernel(float* img, int nx, int ny,
    float* partialReduction)
{
  int k = blockDim.x * blockIdx.x + threadIdx.x;
  int l = blockDim.y * blockIdx.y + threadIdx.y;
  int xdim = (nx/2+1) * 2;
  __shared__ float locRedBuffer[RED_BLOCK_SIZE_X * RED_BLOCK_SIZE_Y];
  if (k < nx && l < ny) {
    locRedBuffer[threadIdx.x + blockDim.x * threadIdx.y] =
      img[k + l * xdim];
  } else {
    locRedBuffer[threadIdx.x + blockDim.x * threadIdx.y] = 0.0f;
  }
  int ltid = threadIdx.y * RED_BLOCK_SIZE_X + threadIdx.x;
  __syncthreads();
  for (int s = RED_BLOCK_SIZE_X * RED_BLOCK_SIZE_Y / 2; s > 0; s >>= 1) {
    if (ltid < s) {
      locRedBuffer[ltid] += locRedBuffer[ltid + s];
    }
    __syncthreads();
  }
  if (ltid == 0) {
    int blockIndex = blockIdx.y * gridDim.x + blockIdx.x;
    partialReduction[blockIndex] = locRedBuffer[0];
  }
}

__global__ void rescale_kernel(float* img, int nx, int ny,
    float scaleFactor)
{
  int k = blockDim.x * blockIdx.x + threadIdx.x;
  int l = blockDim.y * blockIdx.y + threadIdx.y;
  int xdim = (nx/2 + 1) * 2;
  if (k < nx && l < ny) {
    img[l * xdim + k] *= scaleFactor;
  }
}

__host__ void fixdrift_2D(std::vector<GPUBuffer>* CrawImages,
    vector3d *driftlist, int nphases, int nx, int ny, int nz, int dir,
    int z)
{
  printf("In fixdrift_2D.\n");
  fflush(stdout);
}

__host__ int calcRefImage(const std::vector<GPUBuffer>& rawImages,
    GPUBuffer* refImage, const std::vector<GPUBuffer>& offImages,
    int nOffImages, int nx, int ny, int nphases, int type_of_refImage)
{
  printf("In calcRefImage.\n");
  fflush(stdout);
  return 0;
}

__host__ void separate(int nx, int ny, int nz, int direction, int nphases,
    int norders, std::vector<GPUBuffer>*rawImages, float *sepMatrix)
{
#ifndef NDEBUG
  for (auto i = rawImages->begin(); i != rawImages->end(); ++i)
    assert(i->hasNaNs() == false);
#endif
  // Allocate memory for result (have to do this out-of-place)
  std::vector<float*> output(norders * 2 - 1);
  int xdim = (nx/2 + 1) * 2;
  for (auto i = output.begin(); i != output.end(); ++i) {
    cutilSafeCall(hipMalloc((void**)&(*i), nz * ny * xdim *
          sizeof(float)));
  }
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_outputPtrs), &output[0],
        output.size() * sizeof(output[0])));

  // Transfer image pointers in __constant__ array
  std::vector<float*> imgPtrs;
  for (auto i = rawImages->begin(); i != rawImages->end(); ++i) {
    imgPtrs.push_back((float*)i->getPtr());
  }
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_imgPtrs), &imgPtrs[0],
        imgPtrs.size() * sizeof(imgPtrs[0])));

  // Transfer sepMatrix to __constant__ array
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_sepMatrix), &sepMatrix[0],
        (norders * 2 - 1) * nphases * sizeof(sepMatrix[0])));

  // Do the separation step
  int nThreadsX = 16;
  int nThreadsY = 16;
  dim3 nThreads(nThreadsX, nThreadsY, 1);
  int numBlocksX = (int)ceil((float)xdim / nThreadsX);
  int numBlocksY = (int)ceil((float)ny / nThreadsY);
  dim3 nBlocks(numBlocksX, numBlocksY, 1);
  separate_kernel<<<nBlocks, nThreads>>>( norders, nphases, nx, ny, nz);
  cutilSafeCall(hipGetLastError());

  // Release the input data pointers and swap the result pointers into
  // the rawImages
  for (int i = 0; i < nphases; ++i) {
    rawImages->at(i).resize(0);
    rawImages->at(i).setPtr((char*)output[i],
        nz * ny * xdim * sizeof(float), 0);
  }
#ifndef NDEBUG
  for (auto i = rawImages->begin(); i != rawImages->end(); ++i) {
    assert(i->hasNaNs() == false);
  }
#endif
}

__global__ void separate_kernel(int norders, int nphases,
    int nx, int ny, int nz)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y  + threadIdx.y;
  int xdim = (nx/2 + 1) * 2;
  int nxy2 = xdim * ny;
  int offset = y * xdim + x;
  if (x < xdim && y < ny) {
    for (int i = 0; i < norders * 2 - 1; ++i) {
      float* outBasePtr = const_outputPtrs[i];
      for (int z = 0; z < nz; ++z) {
        float result = 0.0f;
        float* outPtr = outBasePtr + z * nxy2;
        for (int j = 0; j < nphases; ++j) {
          const float* imgBasePtr = const_imgPtrs[j];
          const float* imgPtr = imgBasePtr + z * nxy2;
          float mij= const_sepMatrix[i * nphases + j];
          result +=  mij * imgPtr[offset];
        }
        outPtr[offset] = result;
      }
    }
  }
}

__host__ void makemodeldata(int nx, int ny, int nz, std::vector<GPUBuffer>* bands,
    int norders, vector k0, float dy, float dz,
    std::vector<GPUBuffer>* OTF, short wave, ReconParams *pParams) {
  printf("In makemodeldata.\n");
  fflush(stdout);
}

__host__ void fixdrift_bt_dirs(std::vector<GPUBuffer>* bands, int norders,
    vector3d drift, int nx,int ny, int nz) {
  printf("In fixdrift_bt_dirs.\n");
  fflush(stdout);
}

__host__ void findk0(std::vector<GPUBuffer>* bands, GPUBuffer* overlap0,
    GPUBuffer* overlap1, int nx, int ny, int nz, int norders, vector *k0,
    float dxy, float dz, std::vector<GPUBuffer>* OTF, short wave,
    ReconParams * pParams)
{
  int fitorder1;
  int fitorder2;


  fitorder1 = 0;
  if (nz > 1) {
    if (!pParams->bBessel) {
      fitorder2 = 2;
    } else {
      fitorder2 = 1;
    }
  }
  else {
    fitorder2 = 1;
  }

  makeoverlaps(bands, overlap0, overlap1, nx, ny, nz, fitorder1, fitorder2,
      (*k0).x, (*k0).y, dxy, dz, OTF, wave, pParams);

  GPUBuffer crosscorr_c(nx * ny * sizeof(hipFloatComplex), 0);
  aTimesConjB(overlap0, overlap1, nx, ny, nz, &crosscorr_c);

  hipfftHandle cufftplan;
  int err = hipfftPlan2d(&cufftplan, ny, nx, HIPFFT_C2C);
  if (HIPFFT_SUCCESS != err) {
    printf("cufftPlanxd failed at %s(%d)\n", __FILE__, __LINE__);
    printf("Error code: %d\n", err);
    fflush(stdout);
    exit(-1);
  }
  err = hipfftExecC2C(cufftplan, (hipFloatComplex*)crosscorr_c.getPtr(),
      (hipFloatComplex*)crosscorr_c.getPtr(), HIPFFT_FORWARD);
  if (HIPFFT_SUCCESS != err) {
    printf("hipfftExecC2C failed at %s(%d)\n", __FILE__, __LINE__);
    printf("Error code: %d\n", err);
    fflush(stdout);
    exit(-1);
  }
  hipfftDestroy(cufftplan);

  GPUBuffer crosscorr(nx * ny * sizeof(float), 0);
  computeIntensities(&crosscorr_c, nx, ny, &crosscorr);
  CPUBuffer intensitiesHost(crosscorr.getSize());
  crosscorr.set(&intensitiesHost, 0, crosscorr.getSize(), 0);
  //  std::cout << "Cross correlation:" << std::endl;
  //  intensitiesHost.dump(std::cout, nx, 0, nx * ny * sizeof(float));

  float dkx = 1 / (nx*dxy);
  float dky = 1 / (ny*dxy);
  vector old_k0 = *k0;
  findpeak((float*)intensitiesHost.getPtr(), nx, ny, k0);
  //! k0 so far is in pixels
  
  if (old_k0.x/dkx < (*k0).x - nx / 2) (*k0).x -= nx;
  if (old_k0.x/dkx > (*k0).x + nx / 2) (*k0).x += nx;
  if (old_k0.y/dky < (*k0).y - ny / 2) (*k0).y -= ny;
  if (old_k0.y/dky > (*k0).y + ny / 2) (*k0).y += ny;

  // Return k0 in 1/um of the first order, no matter which fitorder2 is used
  k0->x *= dkx / fitorder2;
  k0->y *= dky / fitorder2; 
}

__host__ void aTimesConjB(GPUBuffer* overlap0, GPUBuffer* overlap1,
    int nx, int ny, int nz, GPUBuffer* crosscorr_c)
{
  dim3 blockSize;
  blockSize.x = 16;
  blockSize.y = 16;
  blockSize.z = 1;
  dim3 numBlocks;
  numBlocks.x = (int)(ceil((float)nx / blockSize.x));
  numBlocks.y = (int)(ceil((float)ny / blockSize.y));
  numBlocks.z = 1;
  aTimesConjBKernel<<<numBlocks, blockSize>>>(
      (hipFloatComplex*)overlap0->getPtr(), (hipFloatComplex*)overlap1->getPtr(),
      nx, ny, nz,
      (hipFloatComplex*)crosscorr_c->getPtr());
}

__global__ void aTimesConjBKernel(hipFloatComplex* overlap0,
    hipFloatComplex* overlap1, int nx, int ny, int nz,
    hipFloatComplex* crosscorr_c)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int l = blockIdx.y * blockDim.y + threadIdx.y;
  if (k < nx && l < ny) {
    int nxy = nx * ny;
    overlap0 += l * nx + k;
    overlap1 += l * nx + k;
    hipFloatComplex result;
    result.x = 0.0f;
    result.y = 0.0f;
    for (int z = 0; z < nz; ++z) {
      hipFloatComplex Xval = *overlap0;
      hipFloatComplex Yval = *overlap1;
      result.x += Xval.x * Yval.x + Xval.y * Yval.y;
      result.y += -Xval.x * Yval.y + Xval.y * Yval.x;
      overlap0 += nxy;
      overlap1 += nxy;
    }
    crosscorr_c[l * nx + k] = result;
  }
}

__host__ void computeIntensities(GPUBuffer* amplitudes, int nx, int ny,
    GPUBuffer* intensities)
{
  dim3 blockSize;
  blockSize.x = 16;
  blockSize.y = 16;
  blockSize.z = 1;
  dim3 numBlocks;
  numBlocks.x = (int)(ceil((float)nx / blockSize.x));
  numBlocks.y = (int)(ceil((float)ny / blockSize.y));
  numBlocks.z = 1;
  computeIntensitiesKernel<<<numBlocks, blockSize>>>(
      (hipFloatComplex*)amplitudes->getPtr(), nx, ny,
      (float*)intensities->getPtr());
}

__global__ void computeIntensitiesKernel(hipFloatComplex* amplitudes,
    int nx, int ny, float* intensities)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;
  int l = blockIdx.y * blockDim.y + threadIdx.y;
  if (k < nx && l < ny) {
    hipFloatComplex amp = amplitudes[l * nx + k];
    intensities[l * nx + k] = amp.x * amp.x + amp.y * amp.y;
  }
}

__host__ void findpeak(float array[], int sizex, int sizey, vector *peak)
{
  int   xcent=0, ycent=0, i, j;
  float a1, a2, a3, big;

  big = -1e11;
  for(i=0;i<sizey;i++)
    for(j=0;j<sizex;j++)
      if(array[i*sizex+j] > big) {
        big=array[i*sizex+j];
        ycent = i;  xcent = j;
      }

  if(xcent==0)
    a1 = array[ ycent*sizex +  xcent-1+sizex];
  else
    a1 = array[ ycent*sizex +  xcent-1];
  a2 = array[ ycent*sizex +  xcent  ];
  if(xcent==sizex-1)
    a3 = array[ ycent*sizex +  xcent+1-sizex];
  else
    a3 = array[ ycent*sizex +  xcent+1];
  (*peak).x = fitparabola(a1,a2,a3) + xcent;

  if(ycent==0)
    a1 = array[ (ycent-1+sizey)*sizex + xcent ];
  else
    a1 = array[ (ycent-1)*sizex + xcent ];
  a2 = array[ (ycent  )*sizex + xcent ];
  if(ycent==sizey-1)
    a3 = array[ (ycent+1-sizey)*sizex + xcent ];
  else
    a3 = array[ (ycent+1)*sizex + xcent ];
  (*peak).y = fitparabola(a1,a2,a3) + ycent;
}

__host__ float fitparabola( float a1, float a2, float a3 )
{
  float slope,curve,peak;

  slope = 0.5* (a3-a1);         /* the slope at (x=0).  */
  curve = (a3+a1) - 2*a2;       /* (a3-a2)-(a2-a1). The change in slope per unit of x. */
  if( curve == 0 ) {
    printf("no peak: a1=%f, a2=%f, a3=%f, slope=%f, curvature=%f\n",a1,a2,a3,slope,curve);
    return( 0.0 );
  }
  peak = -slope/curve;          /* the x value where slope = 0  */
  if( peak>1.5 || peak<-1.5 ) {
    printf("bad peak position: a1=%f, a2=%f, a3=%f, slope=%f, curvature=%f, peak=%f\n",a1,a2,a3,slope,curve,peak);
    return( 0.0 );
  }
  return( peak );
}

__host__ void makeoverlaps(std::vector<GPUBuffer>* bands,
    GPUBuffer* overlap0, GPUBuffer* overlap1, int nx, int ny, int nz,
    int order1, int order2, float k0x, float k0y, float dxy, float dz,
    std::vector<GPUBuffer>* OTF, short wave, ReconParams* params)
{
  float order0_2_factor = 1.0f;
  if (nz > 1) {
    order0_2_factor = 5.0f;
    if (params->bBessel)
      order0_2_factor = 4.0f;
  }
  float dkx = 1.0f / (nx * dxy);
  float dky = 1.0f / (ny * dxy);
  float dkz;
  if (dz > 0.0f) {
    dkz = 1.0f / (nz * dz);
  } else {
    dkz = params->dkzotf;
  }

  float kzscale = dkz / params->dkzotf;
  float rdistcutoff = params->na * 2.0 / (wave * 0.001);
  if (rdistcutoff > 1./(2.*dxy) ) rdistcutoff = 1. / (2.*dxy);

  //! k0 magnitude (of the lowest order) in 1/micron
  float k0mag = sqrt(k0x * k0x + k0y * k0y);
  float lambdaem = (wave / params->nimm) / 1.0e3;
  float lambdaexc = 0.88 * lambdaem;
  float alpha = asinf(params->na / params->nimm);
  float beta = asinf(k0mag / (2.0 / lambdaexc));
  float betamin = asinf(k0mag / (2.0 / lambdaexc) - sinf(alpha) *
      SPOTRATIO);
  float zdistcutoff;
  if (!params->bTwolens && !params->bBessel) {
    zdistcutoff = (int)ceil(((1.0 - cosf(alpha)) / lambdaem) / dkz);
  }
  else if (params->bBessel) {
    float halfangle;
    float kzExMax;
    kzExMax = 2.0 * params->BesselNA / params->BesselLambdaEx;
    halfangle = acosf(k0mag * order2 /(params->norders-1)/ kzExMax);
    zdistcutoff = ceil((kzExMax * sinf(halfangle) + (1.0 - cosf(alpha)) / lambdaem) / dkz);
  }
  else {
    std::cerr << "Sorry, this program doesn't handel 2-objective mode data\n";
    exit(-1);
  }


  if (zdistcutoff > nz / 2) {
    zdistcutoff = ((nz / 2 - 1) > 0) ? (nz / 2 - 1) : 0;
  }


  float kx = k0x * (order2 - order1);
  float ky = k0y * (order2 - order1);
  float otfcutoff = params->otfcutoff;
  if (params->bBessel)
    otfcutoff = 0.01;

  cutilSafeCall(hipMemset((void*)overlap0->getPtr(), 0,
        nx * ny * nz * sizeof(hipFloatComplex)));
  cutilSafeCall(hipMemset((void*)overlap1->getPtr(), 0,
        nx * ny * nz * sizeof(hipFloatComplex)));

  // Copy configuration parameters to GPU
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bSuppress_singularities),
        &params->bSuppress_singularities, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_suppression_radius),
        &params->suppression_radius, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bDampenOrder0),
        &params->bDampenOrder0, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bNoKz0),
        &params->bNoKz0, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bFilteroverlaps),
        &params->bFilteroverlaps, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_apodizeoutput),
        &params->apodizeoutput, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bBessel),
        &params->bBessel, sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bRadAvgOTF),
        &params->bRadAvgOTF, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_nzotf),
        &params->nzotf, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_dkrotf), &params->dkrotf,
        sizeof(float)));
  std::vector<hipFloatComplex*> otfPtrs;

  for (int i = 0; i < params->norders; ++i) {
    otfPtrs.push_back((hipFloatComplex*)(OTF->at(i).getPtr()));
    //    OTF->at(i).dump(std::cout, 100, 0, 100 * sizeof(float));
  }
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_otfPtrs),
        &otfPtrs[0], params->norders * sizeof(hipFloatComplex *),
        0, hipMemcpyHostToDevice));


  // Set the band ptrs
  hipFloatComplex* band1re;
  hipFloatComplex* band1im;
  hipFloatComplex* band2re;
  hipFloatComplex* band2im;
  //  std::cout << "bands in makeoverlaps:\n";
  if (order1 == 0) {
    band1re = (hipFloatComplex*)bands->at(0).getPtr();
    band1im = 0;
  } else {
    band1re = (hipFloatComplex*)bands->at(order1 * 2 - 1).getPtr();
    band1im = (hipFloatComplex*)bands->at(order1 * 2).getPtr();
  }
  // It is assumed that order2 is never 0
  band2re = (hipFloatComplex*)bands->at(order2 * 2 - 1).getPtr();
  band2im = (hipFloatComplex*)bands->at(order2 * 2).getPtr();

  // printf("In makeoverlaps(), order1=%d, order2=%d, k0x=%f, k0y=%f, rdistcutoff=%f, zdistcutoff=%f pixels\n", order1, order2, kx/dkx, ky/dky, rdistcutoff/dky, zdistcutoff);
  // Generate the overlap arrays
  int numThreads = 128;
  dim3 threads(numThreads, 1, 1);
  int numBlocksX = nx / numThreads;
  if (nx % numThreads != 0) {
    ++numBlocksX;
  }
  int numBlocksY = ny;
  int numBlocksZ = 2 * (int)zdistcutoff + 1;
  dim3 blocks(numBlocksX, numBlocksY, numBlocksZ);
  makeOverlaps0Kernel<<<blocks,threads>>>(
      nx, ny, nz, order1, order2, kx, ky, rdistcutoff,
      otfcutoff, zdistcutoff, order0_2_factor, dkx, dky, kzscale,
      band1im, band1re, (hipFloatComplex*)overlap0->getPtr());
  cutilSafeCall(hipGetLastError());
  makeOverlaps1Kernel<<<blocks,threads>>>(
      nx, ny, nz, order1, order2, kx, ky, rdistcutoff,
      otfcutoff, zdistcutoff, order0_2_factor, dkx, dky, kzscale,
      band2im, band2re, (hipFloatComplex*)overlap1->getPtr());
  cutilSafeCall(hipGetLastError());

#ifndef NDEBUG
  assert(overlap0->hasNaNs() == false);
  assert(overlap1->hasNaNs() == false);
#endif

  // Do ffts
  hipfftResult err;
  hipfftHandle cufftplan;
  if (nz > 1) {
    err = hipfftPlan3d(&cufftplan, nz, ny, nx, HIPFFT_C2C);
  } else {
    err = hipfftPlan2d(&cufftplan, ny, nx, HIPFFT_C2C);
  }
  if (HIPFFT_SUCCESS != err) {
    printf("cufftPlanxd failed\n");
    printf("Error code: %d\n", err);
    fflush(stdout);
    exit(-1);
  }
  err = hipfftExecC2C(cufftplan, (hipFloatComplex*)overlap0->getPtr(),
      (hipFloatComplex*)overlap0->getPtr(), HIPFFT_BACKWARD);
  if (HIPFFT_SUCCESS != err) {
    printf("hipfftExecC2C failed at %s(%d)\n", __FILE__, __LINE__);
    printf("Error code: %d\n", err);
    fflush(stdout);
    exit(-1);
  }
  err = hipfftExecC2C(cufftplan, (hipFloatComplex*)overlap1->getPtr(),
      (hipFloatComplex*)overlap1->getPtr(), HIPFFT_BACKWARD);
  if (HIPFFT_SUCCESS != err) {
    printf("hipfftExecC2C failed at %s(%d)\n", __FILE__, __LINE__);
    printf("Error code: %d\n", err);
    fflush(stdout);
    exit(-1);
  }

  hipfftDestroy(cufftplan);

#ifndef NDEBUG
  assert(overlap0->hasNaNs() == false);
  assert(overlap1->hasNaNs() == false);
#endif
}

__global__ void makeOverlaps0Kernel(int nx, int ny, int nz,
    int order1, int order2, float kx, float ky,
    float rdistcutoff, float otfcutoff, float zdistcutoff,
    float order0_2_factor, float dkx, float dky, float kzscale,
    hipFloatComplex *band1im, hipFloatComplex *band1re,
    hipFloatComplex *overlap0)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j<nx) {
    int i = blockIdx.y;
    int z0 = blockIdx.z - zdistcutoff;
    float x1f = j;
    float y1f = i;
    if (x1f > nx / 2)
      x1f -= nx;
    if (y1f > ny / 2)
      y1f -= ny;

	// Convert pixels into 1/micron:
	x1f *= dkx;
	y1f *= dky;
    float rdist1 = sqrt(x1f * x1f + y1f * y1f);
    if (rdist1 <= rdistcutoff) {

      float x12 = x1f - kx;
      float y12 = y1f - ky;
      float rdist12 = sqrt(x12 * x12 + y12 * y12);
      float x21 = x1f + kx;
      float y21 = y1f + ky;
      float rdist21 = sqrt(x21 * x21 + y21 * y21);
      if (rdist12 <= rdistcutoff || rdist21 <= rdistcutoff) { 

        int iin;  // coords of input arrays
        int jin;
        int conj;
        if (j <= nx / 2) {
          iin = i;
          jin = j;
          conj = 0;
        } else {
          jin = nx - j;
          iin = (ny - i) % ny;
          conj = 1;
        }

        if (rdist12 <= rdistcutoff) {
          if (!(z0 == 0 && const_pParams_bNoKz0)) {
            hipFloatComplex otf1 = dev_otfinterpolate(
                     const_otfPtrs[order1], x1f, y1f, z0, kzscale);
            if (sqrt(otf1.x * otf1.x + otf1.y * otf1.y) > otfcutoff) {
              hipFloatComplex otf12 = dev_otfinterpolate(
                     const_otfPtrs[order2], x12, y12, z0, kzscale);
              if (sqrt(otf12.x * otf12.x + otf12.y * otf12.y) * order0_2_factor > otfcutoff) {
                int z;
                if (conj) {
                  z = -z0;
                } else {
                  z = z0;
                }
                z = (z + nz) % nz;
                int indin = z * (nx / 2 + 1) * ny + iin * (nx / 2 + 1) + jin;
                hipFloatComplex val1re = band1re[indin];
                hipFloatComplex val1im ={0., 0.};
                if (order1 > 0) {
                  val1im = band1im[indin];
                }
                float root = sqrt(otf1.x * otf1.x + otf1.y * otf1.y +
                                  otf12.x * otf12.x + otf12.y * otf12.y);
                hipFloatComplex fact = otf12;
                fact.x /= root;
                fact.y /= root;
                if (conj) {
                  val1re.y *= -1.0;
                  if (order1 > 0) {
                    val1im.y *= -1.0;
                  }
                }
                float temp = val1re.x * fact.x - val1re.y * fact.y;
                val1re.y = val1re.x * fact.y + val1re.y * fact.x;
                val1re.x = temp;
                if (order1 > 0) {
                  temp = val1im.x * fact.x - val1im.y * fact.y;
                  val1im.y = val1im.x * fact.y + val1im.y * fact.x;
                  val1im.x = temp;
                }

                z = (z0 + nz) % nz;
                int indout = z * nx * ny + i * nx + j;
//              if (order1 > 0) {
                overlap0[indout].x = val1re.x - val1im.y;
                overlap0[indout].y = val1re.y + val1im.x;
                // } else {
                //   overlap0[indout] = val1re;
                // }
              }
            }
          }
        }
      }
    }
  }
}

__global__ void makeOverlaps1Kernel(int nx, int ny, int nz,
    int order1, int order2, float kx, float ky,
    float rdistcutoff, float otfcutoff, float zdistcutoff,
	float order0_2_factor, float dkx, float dky, float kzscale,
    hipFloatComplex *band2im, hipFloatComplex *band2re,
    hipFloatComplex *overlap1)
{
  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j<nx) {
    int i = blockIdx.y;
    int z0 = blockIdx.z - zdistcutoff;
    float x1f = j;
    float y1f = i;
    if (x1f > nx / 2) {
      x1f -= nx;
    }
    if (y1f > ny / 2) {
      y1f -= ny;
    }

	// Convert pixels into 1/micron:
	x1f *= dkx;
	y1f *= dky;
    float rdist1 = sqrt(x1f * x1f + y1f * y1f);
    if (rdist1 < rdistcutoff) {

      float x12 = x1f - kx;
      float y12 = y1f - ky;
      float rdist12 = sqrt(x12 * x12 + y12 * y12);
      float x21 = x1f + kx;
      float y21 = y1f + ky;
      float rdist21 = sqrt(x21 * x21 + y21 * y21);
      if (rdist12 <= rdistcutoff || rdist21 <= rdistcutoff) {

        int iin;
        int jin;
        int conj;
        if (j <= nx / 2) {
          iin = i;
          jin = j;
          conj = 0;
        } else {
          jin = nx - j;
          iin = (ny - i) % ny;
          conj = 1;
        }

        if (rdist21 <= rdistcutoff) {
          if (!(z0 == 0 && const_pParams_bNoKz0)) {
            hipFloatComplex otf2 = dev_otfinterpolate(
                             const_otfPtrs[order2], x1f, y1f, z0, kzscale);
            if (sqrt(otf2.x * otf2.x + otf2.y * otf2.y) * order0_2_factor > otfcutoff) {
              hipFloatComplex otf21 = dev_otfinterpolate(
                               const_otfPtrs[order1], x21, y21, z0, kzscale);
              if (sqrt(otf21.x * otf21.x + otf21.y * otf21.y) > otfcutoff) {
                int z;
                if (conj) {
                  z = -z0;
                } else {
                  z = z0;
                }
                z = (z + nz) % nz;
                int indin = z * (nx / 2 + 1) * ny + iin * (nx / 2 + 1) + jin;
                hipFloatComplex val2re = band2re[indin];
                hipFloatComplex val2im = band2im[indin];
                float root = sqrt(otf2.x * otf2.x + otf2.y * otf2.y +
                                  otf21.x * otf21.x + otf21.y * otf21.y);
                hipFloatComplex fact = otf21;
                fact.x /= root;
                fact.y /= root;
                if (conj) {
                  val2re.y *= -1.0f;
                  val2im.y *= -1.0f;
                }
                float temp = val2re.x * fact.x - val2re.y * fact.y;
                val2re.y = val2re.x * fact.y + val2re.y * fact.x;
                val2re.x = temp;
                temp = val2im.x * fact.x - val2im.y * fact.y;
                val2im.y = val2im.x * fact.y + val2im.y * fact.x;
                val2im.x = temp;

                z = (z0 + nz) % nz;
                int indout = z * nx * ny + i * nx + j;
                overlap1[indout].x = val2re.x - val2im.y;
                overlap1[indout].y = val2re.y + val2im.x;
              }
            }
          }
        }
      }
    }
  }
}

__host__ void fitk0andmodamps(std::vector<GPUBuffer>* bands,
    GPUBuffer* overlap0, GPUBuffer* overlap1, int nx, int ny, int nz,
    int norders, vector *k0, float dxy, float dz, std::vector<GPUBuffer>* otf,
    short wave, hipFloatComplex amps[], ReconParams * pParams)
{ 
  int fitorder1 = 0;
  int fitorder2 = 0;
  if (nz > 1) {
	fitorder2 = 2;
    if (pParams->bBessel) 
      fitorder2 = 1;
  }
  else {
    fitorder2 = 1;
  }

  float k0mag = sqrt(k0->x * k0->x + k0->y * k0->y);  // in 1/um
  float k0angle = atan2(k0->y, k0->x);

  /* recalculate the overlap arrays at least this first time */
  int redoarrays = (pParams->recalcarrays >= 1);
  float x2 = k0angle;
  hipFloatComplex modamp;
  float amp2 = getmodamp(k0angle, k0mag, bands, overlap0,  overlap1, nx, ny, nz,
      fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);

  /* recalculate the overlap arrays every time only if recalcarrays >= 3*/
  redoarrays = (pParams->recalcarrays >= 3);
  float deltaangle = 0.001;
  float deltamag = 0.1 / (std::max(nx, ny) * dxy);  // in 1/um
  float angle = k0angle + deltaangle;
  float x3 = angle;
  float amp3 = getmodamp(angle, k0mag, bands, overlap0,  overlap1, nx, ny, nz,
      fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);

  float amp1;
  float x1 = 0.0;
  float a;
  if (amp3 > amp2) {
    while(amp3 > amp2) {
      amp1 = amp2;
      x1 = x2;
      amp2 = amp3;
      x2 = x3;
      angle += deltaangle;
      x3 = angle;
      amp3 = getmodamp(angle, k0mag, bands, overlap0, overlap1, nx, ny, nz,
          fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);
    }
  } else {
    angle = k0angle;
    a = amp3;
    amp3 = amp2;
    amp2 = a;
    a = x3;
    x3 = x2;
    x2 = a;
    while (amp3 > amp2) {
      amp1 = amp2;
      x1 = x2;
      amp2 = amp3;
      x2 = x3;
      angle -= deltaangle;
      x3 = angle;
      amp3 = getmodamp(angle, k0mag, bands, overlap0, overlap1, nx, ny, nz,
          fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);
    }
  }  /* the maximum of modamp(x) is now between x1 and x3 */
  angle = fitxyparabola(x1, amp1, x2, amp2, x3, amp3);   /* this should be a good angle.  */

  /***** now search for optimum magnitude, at this angle  *****/

  x2 = k0mag;
  amp2 = getmodamp(angle, k0mag, bands, overlap0, overlap1, nx, ny, nz,
      fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);

  float mag = k0mag + deltamag;
  x3 = mag;
  amp3 = getmodamp(angle, mag, bands, overlap0, overlap1, nx, ny, nz,
      fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);
  if (amp3 > amp2) {
    while (amp3 > amp2) {
      amp1 = amp2;
      x1 = x2;
      amp2 = amp3;
      x2 = x3;
      mag += deltamag;
      x3 = mag;
      amp3 = getmodamp(angle, mag, bands, overlap0, overlap1, nx, ny, nz,
          fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);
    }
  } else {
    mag = k0mag;
    a = amp3;
    amp3 = amp2;
    amp2 = a;
    a = x3;
    x3 = x2;
    x2 = a;
    while (amp3 > amp2) {
      amp1 = amp2;
      x1 = x2;
      amp2 = amp3;
      x2 = x3;
      mag -= deltamag;
      x3 = mag;
      amp3 = getmodamp(angle, mag, bands, overlap0, overlap1, nx, ny, nz,
          fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 0);
    }
  }  /* the maximum of modamp(x) is now between x1 and x3 */

  mag = fitxyparabola(x1, amp1, x2, amp2, x3, amp3);  /* this should be a good magnitude.  */

  /* if we were perfectionist we'd iterate for angle again now */

  printf("Optimum modulation amplitude:\n");
  redoarrays = (pParams->recalcarrays>=2);    /* recalculate the d_overlap arrays for optimum modamp fit */
  amp3 = getmodamp(angle, mag, bands, overlap0,  overlap1, nx, ny, nz,
      fitorder1, fitorder2, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 1);
  /* one last time, to find the modamp at the optimum k0*/

  printf("Optimum k0 angle=%f, length=%f, spacing=%f um\n", angle, mag, 1.0 / mag);

  k0->x = mag * cosf(angle);
  k0->y = mag * sinf(angle);
  amps[fitorder2] = modamp;

  /* finally find the modamp for the other orders */
  redoarrays=1;
  if (nz == 1) {
    for (int order = 2; order < norders; ++order) {
      /* assuming that "angle" and "mag" remain the same for every adjacent pair of bands within one direction */
      getmodamp(angle, mag, bands, overlap0, overlap1, nx, ny, nz,
          order - 1, order, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 1);
      amps[order] = modamp;
    }
  } else {
    /* 3D */
    for (int order = 1; order < norders; ++order) {
      if (order != fitorder2) {
        getmodamp(angle, mag, bands, overlap0, overlap1, nx, ny, nz,
            0, order, dxy, dz, otf, wave, &modamp, redoarrays, pParams, 1);
        amps[order] = modamp;
      }
    }
  }
}

__host__ float fitxyparabola( float x1, float y1, float x2, float y2, float x3, float y3 )
{
  float slope1,slope2,curve,peak,xbar1,xbar2;

  if( x1==x2 || x2==x3 || x3==x1 ) {
    printf("Fit fails; two points are equal: x1=%f, x2=%f, x3=%f\n",x1,x2,x3);
    return( 0.0 );
  }
  xbar1 = 0.5 * (x1 + x2);               /* middle of x1 and x2 */
  xbar2 = 0.5 * (x2 + x3);               /* middle of x2 and x3 */
  slope1 = (y2-y1)/(x2-x1);    /* the slope at (x=xbar1).  */
  slope2 = (y3-y2)/(x3-x2);    /* the slope at (x=xbar2).  */
  curve = (slope2-slope1) / (xbar2-xbar1);       /* The change in slope per unit of x. */
  if( curve == 0 ) {
    printf("Fit fails; no curvature: r1=(%f,%f), r2=(%f,%f), r3=(%f,%f) slope1=%f, slope2=%f, curvature=%f\n",
        x1,y1,x2,y2,x3,y3, slope1,slope2,curve);
    return( 0.0 );
  }

  peak = xbar2 - slope2/curve;          /* the x value where slope = 0  */

  return( peak );
}

__host__ float getmodamp(float kangle, float klength,
    std::vector<GPUBuffer>* bands, GPUBuffer* overlap0, GPUBuffer* overlap1,
    int nx, int ny,int nz, int order1, int order2, float dxy, float dz,
    std::vector<GPUBuffer>* otf, short wave, hipFloatComplex* modamp,
    int redoarrays, ReconParams *pParams, int bShowDetail)
{
  vector k1;
  float amp2;
  float corr_coef;
  hipFloatComplex amp_inv;
  hipFloatComplex amp_combo;

  k1.x = klength * cosf(kangle);
  k1.y = klength * sinf(kangle);
  corr_coef = findrealspacemodamp(bands, overlap0, overlap1, nx, ny, nz, order1, order2,
	  k1, dxy, dz, otf, wave, modamp, &amp_inv, &amp_combo, redoarrays, pParams);
  amp2 = modamp->x * modamp->x + modamp->y * modamp->y;

  printf(" In getmodamp: angle=%f, mag=%f, amp=%f, phase=%f\n", kangle, klength, sqrt(amp2), get_phase(*modamp));
  if (bShowDetail) {
    printf(" Reverse modamp is: amp=%f, phase=%f\n", 1.0 / cmag(amp_inv), -get_phase(amp_inv));
    printf(" Combined modamp is: amp=%f, phase=%f\n", cmag(amp_combo), get_phase(amp_combo));
    printf(" Correlation coefficient is: %f\n", corr_coef);
  }

  return(amp2);
}

__host__ float findrealspacemodamp(
    std::vector<GPUBuffer>* bands,
    GPUBuffer* overlap0, GPUBuffer* overlap1,
    int nx, int ny, int nz, int order1, int order2,
    vector k0, float dxy, float dz,
    std::vector<GPUBuffer>* OTF,
    short wave,
    hipFloatComplex *modamp1, hipFloatComplex *modamp2,
    hipFloatComplex *modamp3, int redoarrays,
    ReconParams *pParams)
{
  if (redoarrays) {
    /* make arrays that contain only the overlapping parts of fourier
       space. Otf-equalize there, set to zero elsewhere  */
    makeoverlaps(bands, overlap0, overlap1, nx, ny, nz, order1, order2,
        k0.x, k0.y, dxy, dz, OTF, wave, pParams);
  }

  // Launch reduction kernel
  float k0x = k0.x * (order2 - order1);
  float k0y = k0.y * (order2 - order1);

  int numRedBlocksX = (int)ceil((float)nx / (float)RED_BLOCK_SIZE_X);
  int numRedBlocksY = (int)ceil((float)ny / (float)RED_BLOCK_SIZE_Y);
  int numRed = numRedBlocksX * numRedBlocksY;
  GPUBuffer XStarY_dev(numRed * sizeof(hipFloatComplex), 0);
  GPUBuffer sumXMag_dev(numRed * sizeof(float), 0);
  GPUBuffer sumYMag_dev(numRed * sizeof(float), 0);
  CPUBuffer XStarY(numRed * sizeof(hipFloatComplex));
  CPUBuffer sumXMag(numRed * sizeof(float));
  CPUBuffer sumYMag(numRed * sizeof(float));

  int numBlocksX = (int)ceil((float)nx / (float)RED_BLOCK_SIZE_X);
  int numBlocksY = (int)ceil((float)ny / (float)RED_BLOCK_SIZE_Y);
  dim3 blocks(numBlocksX, numBlocksY, 1);
  dim3 threads(RED_BLOCK_SIZE_X, RED_BLOCK_SIZE_Y, 1);
  reductionKernel<<<blocks,threads>>>(nx, ny, nz, k0x, k0y, dxy,
      (hipFloatComplex*)overlap0->getPtr(),
      (hipFloatComplex*)overlap1->getPtr(),
      (hipFloatComplex*)XStarY_dev.getPtr(),
      (float*)sumXMag_dev.getPtr(),
      (float*)sumYMag_dev.getPtr());

  // Get partially reduced data
  XStarY_dev.set(&XStarY, 0, XStarY_dev.getSize(), 0);
  sumXMag_dev.set(&sumXMag, 0, sumXMag_dev.getSize(), 0);
  sumYMag_dev.set(&sumYMag, 0, sumYMag_dev.getSize(), 0);

  // Do rest of reduction
  hipFloatComplex XStarYFR = cpuReduce((hipFloatComplex*)XStarY.getPtr(), numRed);
  float sumXMagFR = cpuReduce((float*)sumXMag.getPtr(), numRed);
  float sumYMagFR = cpuReduce((float*)sumYMag.getPtr(), numRed);

  // Compute results
  modamp1->x = XStarYFR.x / sumXMagFR;
  modamp1->y = XStarYFR.y / sumXMagFR;
  modamp2->x = XStarYFR.x / sumYMagFR;
  modamp2->y = -XStarYFR.y / sumYMagFR;
  float tan2beta = 2.0f * sqrt(XStarYFR.x * XStarYFR.x +
      XStarYFR.y * XStarYFR.y) / (sumXMagFR - sumYMagFR);
  float beta = 0.5f * atan(tan2beta);
  if (beta < 0.0f) {
    beta += 0.5f * M_PI;
  }
  float modamp_amp = tan(beta);
  float modamp_arg = atan2(XStarYFR.y, XStarYFR.x);
  modamp3->x = modamp_amp * cosf(modamp_arg);
  modamp3->y = modamp_amp * sinf(modamp_arg);

  float corr_coef = (XStarYFR.x * XStarYFR.x + XStarYFR.y * XStarYFR.y)
    / (sumXMagFR * sumYMagFR);
  corr_coef = sqrt(corr_coef);

  return corr_coef;
}

__global__ void reductionKernel(
    int nx, int ny, int nz,
    float k0x, float k0y, float dxy,
    const hipFloatComplex *overlap0, const hipFloatComplex *overlap1,
    hipFloatComplex *XStarY, float *sumXMag, float *sumYMag) {

  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.y + blockIdx.y * blockDim.y;
  int strideZ = nx * ny;

  // phase factor
  hipFloatComplex expiphi;
  float angle = 2.0f * M_PI * (
      ((float)i - 0.5f * (float)nx) * k0x +
      ((float)j - 0.5f * (float)ny) * k0y ) * dxy;
  sincosf(angle, &(expiphi.y), &(expiphi.x));

  // reduction at thread level: each thread sums over z dimension
  hipFloatComplex xsy;
  xsy.x = 0.0f;
  xsy.y = 0.0f;
  float sxm = 0.0f;
  float sym = 0.0f;
  overlap0 += j * nx + i;
  overlap1 += j * nx + i;
  hipFloatComplex czero;
  czero.x = 0.0f;
  czero.y = 0.0f;
  for (int z = 0; z < nz; ++z) {
    hipFloatComplex Xval =
      (i < nx && j < ny) ? overlap0[z * strideZ] : czero;
    hipFloatComplex Yval =
      (i < nx && j < ny) ? overlap1[z * strideZ] : czero;
    xsy.x += Xval.x * Yval.x + Xval.y * Yval.y;
    xsy.y += Xval.x * Yval.y - Xval.y * Yval.x;
    sxm += Xval.x * Xval.x + Xval.y * Xval.y;
    sym += Yval.x * Yval.x + Yval.y * Yval.y;
  }
  float temp = xsy.x * expiphi.x - xsy.y * expiphi.y;
  xsy.y = xsy.x * expiphi.y + xsy.y * expiphi.x;
  xsy.x = temp;

  // reduction at thread block level: summation over tile in x-y plane
  // this is a pretty naive implementation.
  __shared__ hipFloatComplex xsyShrd[RED_BLOCK_SIZE_Y * RED_BLOCK_SIZE_X];
  __shared__ float sxmShrd[RED_BLOCK_SIZE_Y * RED_BLOCK_SIZE_X]; 
  __shared__ float symShrd[RED_BLOCK_SIZE_Y * RED_BLOCK_SIZE_X]; 
  int ltid = threadIdx.y * RED_BLOCK_SIZE_X + threadIdx.x; 
  xsyShrd[ltid] = xsy;
  sxmShrd[ltid] = sxm; 
  symShrd[ltid] = sym; 
  __syncthreads();
  int s;
  for (s = RED_BLOCK_SIZE_Y * RED_BLOCK_SIZE_X / 2; s > 0; s >>= 1) {
    if (ltid < s) {
      xsyShrd[ltid].x += xsyShrd[ltid + s].x;
      xsyShrd[ltid].y += xsyShrd[ltid + s].y;
      sxmShrd[ltid] += sxmShrd[ltid + s];
      symShrd[ltid] += symShrd[ltid + s];
    }
    __syncthreads();
  }

  // write result back
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    int blockIndex = blockIdx.y * gridDim.x + blockIdx.x;
    XStarY[blockIndex] = xsyShrd[0];
    sumXMag[blockIndex] = sxmShrd[0];
    sumYMag[blockIndex] = symShrd[0];
  }
}

__host__ void filterbands(int dir, std::vector<GPUBuffer>* bands,
    const std::vector<vector>& k0, int ndirs, int norders,
    std::vector<GPUBuffer>& otf, float dxy, float dz,
    const std::vector<std::vector<hipFloatComplex> >& amp,
    const std::vector<float>& noiseVarFactors, int nx, int ny, int nz,
    short wave, ReconParams* pParams)
{
  int order, order2, dir2;
  float dkz;
  float lambdaem, lambdaexc, alpha, beta, betamin, wiener;

  wiener = pParams->wiener*pParams->wiener;

  if (dz>0)
    dkz = (1/(nz*dz));   /* inverse microns per pixel in data */
  else
    dkz = pParams->dkzotf;

  float kzscale = dkz / pParams->dkzotf;   /* ratio of axial direction pixel scales of data and otf */
  float k0mag =  sqrt(k0[0].x*k0[0].x + k0[0].y*k0[0].y); //! k0 magnitude (for highest order) in 1/um

  lambdaem = (wave/pParams->nimm)/1000.0;  /* emission wavelength in the sample, in microns */
  lambdaexc = 0.88* lambdaem;;  /* 0.88 approximates a typical lambdaexc/lambdaem  */
  alpha = asinf(pParams->na/pParams->nimm);  /* aperture angle of objectives */
  beta = asinf(k0mag/(2/lambdaexc));   /* angle of center of side illumination beams */
  betamin = asinf((k0mag/(2/lambdaexc)) -sinf(alpha)*SPOTRATIO);   /* angle of inner edge of side illumination beams */
  float rdistcutoff = (pParams->na*2/(wave/1000.0)); /* OTF support radial limit in 1/um */
  if (rdistcutoff> 1./(2.*dxy)) rdistcutoff = 1./(2.*dxy);

  /* 080201: zdistcutoff[0] depends on options -- single or double lenses */
  int * zdistcutoff = (int *) malloc(norders * sizeof(int));
  if (!pParams->bTwolens && !pParams->bBessel) {
    zdistcutoff[0] = (int) ceil(((1-cosf(alpha))/lambdaem) / dkz);    /* OTF support axial limit in data pixels */
    zdistcutoff[norders-1] = 1.3*zdistcutoff[0];    /* approx max axial support limit of the OTF of the high frequency side band */
    if (norders>=3)
      for (order=1;order<norders-1;order++)
        zdistcutoff[order] = (1+lambdaem/lambdaexc)*zdistcutoff[0];       /* axial support limit of the OTF of the medium frequency side band(s?) */
  }
  else if (pParams->bBessel) {
    float kzExMax, halfangle;
    kzExMax = 2 *pParams->BesselNA / pParams->BesselLambdaEx;

    zdistcutoff[0] = (int) rint((kzExMax + (1-cosf(alpha))/lambdaem) / dkz);    /* OTF support axial limit in data pixels */
    printf("norders=%d, zdistcutoff[%d]=%d\n", norders, 0, zdistcutoff[0]);
    for (order=1; order<norders; order++) {
      halfangle = acosf(k0mag * order / (norders-1) / kzExMax);
      zdistcutoff[order] = ceil((kzExMax * sinf(halfangle) + (1.0 - cosf(alpha)) / lambdaem) / dkz);
      printf("zdistcutoff[%d]=%d\n", order ,zdistcutoff[order]);
    }
  }
  else {  /* two lenses */
    zdistcutoff[0] = (int) ceil(1.02*(2/lambdaem + 2/lambdaexc) / dkz);  /* 1.02 is just a safety margin */
    zdistcutoff[norders-1] = (int) ceil(1.02*(2/lambdaem + 2*cosf(beta)/lambdaexc) / dkz);    /* approx max axial support limit of the OTF of the high frequency side band */
    if (norders==3) {
      zdistcutoff[1] =  (int) ceil(1.02*(2/lambdaem + (1+cosf(betamin))/lambdaexc) / dkz); /* axial support limit of the OTF of the medium frequency side band */
    }
    else if (norders>3)
      for (order=1;order<norders-1;order++) {
        float a;
        a = ((float)order)/(norders-1);
        zdistcutoff[order] = 1.1*((1-a)*zdistcutoff[0] + a*zdistcutoff[norders-1]);       /* axial support limit of the OTF of the medium frequency side bands */ /* 1.1 is a blur margin */
      }
  }

  for (order=0;order<norders;order++) {
    if (zdistcutoff[order]>=nz/2) zdistcutoff[order]=((nz/2-1) > 0 ? (nz/2-1) : 0);
    // printf("order=%d, rdistcutoff=%f, zdistcutoff=%d\n", order, rdistcutoff, zdistcutoff[order]);
  }

  float apocutoff = rdistcutoff+ k0mag * (norders-1);

  float zapocutoff;
  if (pParams->bTwolens || pParams->bBessel)
    zapocutoff = zdistcutoff[0];
  else
    zapocutoff = zdistcutoff[1];

  float *ampmag2 = (float *) malloc(norders * sizeof(float));
  hipFloatComplex *conjamp = (hipFloatComplex *) malloc(norders * sizeof(hipFloatComplex));
  for (order=0;order<norders;order++) {
    ampmag2[order] = amp[dir][order].x * amp[dir][order].x +
      amp[dir][order].y * amp[dir][order].y;
    conjamp[order] = amp[dir][order];
    conjamp[order].y *= -1;
  }

  /////////////////////////////////////
  // Move Data into Constant memory
  /////////////////////////////////////
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bSuppress_singularities),
        &pParams->bSuppress_singularities, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_suppression_radius),
        &pParams->suppression_radius, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bDampenOrder0),
        &pParams->bDampenOrder0, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bNoKz0), &pParams->bNoKz0,
        sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bFilteroverlaps),
        &pParams->bFilteroverlaps, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_apodizeoutput),
        &pParams->apodizeoutput, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_apoGamma),
        &pParams->apoGamma, sizeof(float)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bBessel), &pParams->bBessel, sizeof(bool)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_bRadAvgOTF),
        &pParams->bRadAvgOTF, sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_nzotf), &pParams->nzotf,
        sizeof(int)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_pParams_dkrotf), &pParams->dkrotf,
        sizeof(float)));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_wiener), &wiener,
        sizeof(float)));

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_zdistcutoff), zdistcutoff,
        norders*sizeof(int), 0, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_ampmag2), ampmag2,
        norders * sizeof(float), 0, hipMemcpyHostToDevice));

  // Explicitly calculate mag2 of amp for all orders
  float * ampmag2_alldirs = (float *) malloc(ndirs*norders*sizeof(float));
  for (dir2=0; dir2<ndirs; dir2++) {
    for (order2=0;order2<norders;order2++) {
      ampmag2_alldirs[dir2*norders+order2] =
        amp[dir2][order2].x * amp[dir2][order2].x +amp[dir2][order2].y * amp[dir2][order2].y;
    }
  }
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_ampmag2_alldirs),
        ampmag2_alldirs, ndirs* norders * sizeof(float), 0,
        hipMemcpyHostToDevice));
  free(ampmag2_alldirs);

  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_conjamp), conjamp,
        norders * sizeof(hipFloatComplex), 0, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_noiseVarFactors),
        &noiseVarFactors[0], ndirs* norders * sizeof(float),
        0, hipMemcpyHostToDevice));
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_k0), &k0[0],
        ndirs * sizeof(float2),
        0, hipMemcpyHostToDevice));
  //  DM 13/12/2012: The OTFs look fine at this point.
  //  dumpBands(&otf, 128, 257, 1);
  //  exit(0);
  std::vector<hipFloatComplex*> otfPtrs;
  for (auto i = otf.begin(); i != otf.end(); ++i) {
    otfPtrs.push_back((hipFloatComplex*)i->getPtr());
  }
  cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(const_otfPtrs), &otfPtrs[0], 
        norders*sizeof(hipFloatComplex *), 0, hipMemcpyHostToDevice));

#ifndef NDEBUG
  ///////////////////////////////////////////////////////
  // Allocate space for temporary buffers
  ///////////////////////////////////////////////////////
  hipFloatComplex *dev_tempbandplus; // hipFloatComplex array of size nx*ny*nz
  hipFloatComplex *dev_scale;        // hipFloatComplex array of size nx*ny*nz
  cutilSafeCall(hipMalloc((void**) &dev_tempbandplus,
        nx*ny*nz*sizeof(hipFloatComplex)));
  cutilSafeCall(hipMemset((void*) dev_tempbandplus, 0,
        nx*ny*nz*sizeof(hipFloatComplex)));

  cutilSafeCall(hipMalloc((void**) &dev_scale, nx*ny*nz*sizeof(hipFloatComplex)));
  cutilSafeCall(hipMemset((void*) dev_scale, 0, nx*ny*nz*sizeof(hipFloatComplex)));
#endif

  hipFloatComplex * dev_bandptr, * dev_bandptr2;
  for (order=0;order<norders;order++) {
    if (order==0) {
      dev_bandptr = (hipFloatComplex*)bands->at(0).getPtr();
    }
    else {
      dev_bandptr = (hipFloatComplex*)bands->at(2*order-1).getPtr();
      dev_bandptr2 = (hipFloatComplex*)bands->at(2*order).getPtr();
    }


    //
    // KERNEL 1
    //
    int nThreads = 128;
    int NZblock = 2*zdistcutoff[order]+1;
    int NYblock = ny;
    int NXblock = (int) ceil( (float)nx/2./nThreads );
    dim3 grid(NXblock, NYblock, NZblock);
    dim3 block(nThreads, 1, 1);

    filterbands_kernel1<<<grid,block>>>(dir, ndirs, order, norders, nx, ny, nz,
										rdistcutoff, zapocutoff, apocutoff, dxy, kzscale,
										dev_bandptr, dev_bandptr2, false);
    cutilSafeCall(hipGetLastError());

    filterbands_kernel1<<<grid,block>>>(dir, ndirs, order, norders, nx, ny, nz,
										rdistcutoff, zapocutoff, apocutoff, dxy,kzscale,
										dev_bandptr, dev_bandptr2, true);
    cutilSafeCall(hipGetLastError());

#ifndef NDEBUG
    CPUBuffer dev_scale_debug(nx * ny * nz * sizeof(hipFloatComplex));
    hipMemcpy(dev_scale_debug.getPtr(), dev_scale, nx * ny
        * nz * sizeof(hipFloatComplex), hipMemcpyDeviceToHost);
    assert(dev_scale_debug.hasNaNs() == false);
#endif

    //
    // KERNEL 3
    //
    if ((nz-zdistcutoff[order]) > (zdistcutoff[order]+1)) {
      NZblock = (nz-zdistcutoff[order]) - (zdistcutoff[order]+1);
      NXblock = (int) ceil( ((float)(nx/2+1))/nThreads );
      dim3 grid2(NXblock, NYblock, NZblock);
      filterbands_kernel3<<<grid2,block>>>(order, nx, ny, nz, dev_bandptr, dev_bandptr2);
      cutilSafeCall(hipGetLastError());
    }

  } /* for order */

  free(zdistcutoff);
  free(ampmag2);
  free(conjamp);
  return;
}

__global__ void filterbands_kernel1(int dir, int ndirs, int order, int norders, int nx, int ny, 
									int nz, float rdistcutoff, float zapocutoff, float apocutoff, float dxy, float kzscale,
									hipFloatComplex * dev_bandptr, hipFloatComplex * dev_bandptr2, bool bSecondEntry)
{

  float kx, ky, rdist1, rdistabs, apofact;
  kx = order * const_k0[dir].x;
  ky = order * const_k0[dir].y;

  float dkx = 1./(nx*dxy);
  float dky = 1./(ny*dxy);
  float min_dkr = dkx < dky ? dkx : dky;
  float suppRadius = const_pParams_suppression_radius * min_dkr;

  // compute x1, y1, z0 based on block and thread indices
  int x1 = blockIdx.x * blockDim.x + threadIdx.x + 1;
  if (x1 <= nx/2) {

    if (bSecondEntry)
      x1 -= nx/2;
    int y1 = blockIdx.y - ny/2;
    int z0 = blockIdx.z - const_zdistcutoff[order];

    float xabs, yabs;
    int iin, jin, conj, xyind, ind, iz, z;
    hipFloatComplex scale, bandreval, bandimval, bandplusval, bandminusval;

    hipFloatComplex otf1, otf2;
    float weight, sumweight, dampfact;
  
    /*x1, y1 are coords within each band to be scaled */
    if (x1 >= 0) {
      /* integer coords of actual arrays to be filtered */
      iin = y1; 
      jin = x1;
      conj = 0;
    } else {
      iin = -y1;
      jin = -x1;
      conj = 1;
    }

    if (order==0 && conj) return;   /* For center band only the actual pixels need to be filtered */

    if (iin<0) {
      iin += ny;
    }
    xyind = iin * (nx / 2 + 1) + jin;

	float x1f = x1 * dkx;
	float y1f = y1 * dky;
    rdist1 = sqrtf(x1f*x1f+y1f*y1f);  /* dist from center of band to be filtered */

    // The following variable masks the whole computation
    if (rdist1<=rdistcutoff) { /* is x1,y1 within the theoretical lateral OTF support of 
                                  the data that is to be scaled? */
      xabs=x1f+kx;   /* (floating point) coords rel. to absolute fourier space, with */
      yabs=y1f+ky;   /* the absolute origin=(0,0) after the band is shifted by k0 */
      rdistabs = sqrt(xabs*xabs + yabs*yabs);  // used later for apodization calculation
      otf1 = dev_otfinterpolate(const_otfPtrs[order], x1f, y1f, z0, kzscale);

      weight = otf1.x * otf1.x + otf1.y * otf1.y;
      if (order!= 0) weight *= const_ampmag2[order];
      dampfact = 1. / const_noiseVarFactors[dir*norders+order];
    
      // this one is thread dependent ... from the rdist calculation
      if (const_pParams_bSuppress_singularities && order != 0 && rdist1 <=suppRadius)
        dampfact *= dev_suppress(rdist1/min_dkr);
    
      // these next two are not thread dependent
      else if (!const_pParams_bDampenOrder0 && const_pParams_bSuppress_singularities && order ==0  && rdist1 <=suppRadius)
        dampfact *= dev_suppress(rdist1/min_dkr);
    
      else if (const_pParams_bDampenOrder0 && order ==0)
        dampfact *= dev_order0damping(rdist1, z0, rdistcutoff, const_zdistcutoff[0]);
    
      // if no kz=0 plane is used:
      if (order==0 && z0==0 && const_pParams_bNoKz0) dampfact = 0;
    
      weight *= dampfact;
      sumweight=weight;

      int dir2, order2;
      float amp2mag2, kx2, ky2, rdist2;
      float x2, y2;
      for (dir2=0; dir2<ndirs; dir2++) {
        for (order2=-(norders-1); order2<norders; order2++) {
          if (dir2==dir && order2==order) continue;
          if (!const_pParams_bFilteroverlaps && !(order2==0 && order==0)) continue; /* bFilteroverlaps is always true except when (during debug) generating an unfiltered exploded view */
          amp2mag2 = const_ampmag2_alldirs[dir2*norders+abs(order2)];
          kx2 = order2 * const_k0[dir2].x;
          ky2 = order2 * const_k0[dir2].y;
          x2 = xabs-kx2; /* coords rel to shifted center of band 2 */
          y2 = yabs-ky2;
          rdist2 = sqrt(x2*x2+y2*y2);       /* dist from center of band 2 */
    
          if (rdist2<rdistcutoff) {
      
            otf2 = dev_otfinterpolate(const_otfPtrs[abs(order2)], x2, y2, z0, kzscale);
            weight = dev_mag2(otf2) / const_noiseVarFactors[dir2*norders+abs(order2)];
            if (order2 != 0) weight *= amp2mag2;
      
            if (const_pParams_bSuppress_singularities && order2 != 0 && rdist2 <= suppRadius)
              weight *= dev_suppress(rdist2/min_dkr);
      
            else if (!const_pParams_bDampenOrder0 && const_pParams_bSuppress_singularities && order2 ==0 &&  rdist2 <= suppRadius)
              weight *= dev_suppress(rdist2/min_dkr);
      
            else if (const_pParams_bDampenOrder0 && order2==0)
              weight *= dev_order0damping(rdist2, z0, rdistcutoff, const_zdistcutoff[0]);
      
            if (const_pParams_bNoKz0 && order2==0 && z0==0) weight = 0.0f;
      
            sumweight += weight;
          }
        }
      }
  
      sumweight += const_wiener;
      scale.x = dampfact *   otf1.x/sumweight;
      scale.y = dampfact * (-otf1.y)/sumweight;

      if (const_pParams_apodizeoutput) {
        float rho, zdistabs;
        zdistabs = abs(z0);

        if (zapocutoff > 0) {  /* 3D case */
          if (!const_pParams_bBessel)
			rho = sqrt((rdistabs / apocutoff) * (rdistabs / apocutoff) +
					   (zdistabs / zapocutoff) * (zdistabs / zapocutoff));
		  else {
            float rhox, rhoy, rhoz;
            rhox = xabs/rdistcutoff; //apocutoff * 1.6f;
            rhoy = yabs/apocutoff;
            rhoz = zdistabs/zapocutoff;
            rho = sqrt(rhox*rhox + rhoy*rhoy + rhoz*rhoz);
          }
        }
        else         /* 2D case */
          rho = sqrt((rdistabs/apocutoff)*(rdistabs/apocutoff));

        if (rho > 1.f) rho = 1.0f;

        if (const_pParams_apodizeoutput == 1)    /* cosine-apodize */
          apofact = cosf((M_PI*0.5f)* rho);
        else if (const_pParams_apodizeoutput == 2)
          apofact = 1.0f - rho;
        // apofact = __powf(1.0f - rho, const_pParams_apoGamma);
        scale.x *= apofact;
        scale.y *= apofact;
      }
      /* What we want is to use mag2 for the weights, as you have done, and
       * then set  scale = conjugate(otf1)/sumweight */

      /* separate (for this pixel) the even and odd "bands" into the true
       * plus and minus bands */
      /* apply the scale to the plus band only (this will apply it to the
       * minus band as well by symmetry?) */
      /* reassemble into the even and odd "bands" */
      if (conj) {
        z = -z0;
      } else {
        z = z0;
      }
      /* coords of the fourier space arrays have the origin of fourier space
       * at (0,0,0) */
      iz = (z + nz) % nz;
      /* index of the corresponding point in the input array */
      ind = iz*((nx/2+1)*ny) + xyind;
      if (order == 0) {
        // if (!conj) // Condition "order == 0 && conj" has been ruled out earlier
        dev_bandptr[ind] = hipCmulf(dev_bandptr[ind], scale);
      }
      else {
        scale = hipCmulf(scale, const_conjamp[order]); /* not invamp: the 1/|amp| factor is
                                                         taken care of by including ampmag2 in the weights */
        bandreval = dev_bandptr[ind];
        bandimval = dev_bandptr2[ind];
        if (conj) {
          bandreval.y *= -1.0f;
          bandimval.y *= -1.0f;
        }
        /* bandplus = bandre + i bandim */
        bandplusval.x = bandreval.x - bandimval.y;
        bandplusval.y = bandreval.y + bandimval.x;
        /* bandminus = bandre - i bandim */
        bandminusval.x = bandreval.x + bandimval.y;
        bandminusval.y = bandreval.y - bandimval.x;
        /* scale only the bandplus part - bandminus will take care 
           of itself because of symmetry (?) */
        bandplusval = hipCmulf(bandplusval, scale);

        bandreval.x = 0.5f*( bandplusval.x + bandminusval.x);
        bandreval.y = 0.5f*( bandplusval.y + bandminusval.y);
        bandimval.x = 0.5f*( bandplusval.y - bandminusval.y);
        bandimval.y = 0.5f*(-bandplusval.x + bandminusval.x);
        if (conj) {
          bandreval.y *= -1.f;
          bandimval.y *= -1.f;
        }
        dev_bandptr[ind] = bandreval;
        dev_bandptr2[ind] = bandimval;
      }
    }
    else { //if (rdisk1>...)
      iz = (z0 + nz) % nz;
      ind = iz * ((nx / 2 + 1) * ny) + xyind;
      dev_bandptr[ind] = make_hipFloatComplex(0.f, 0.f);
      if (order != 0)
        dev_bandptr2[ind] = make_hipFloatComplex(0.f, 0.f);
    }
  }
  return;
}



__global__ void filterbands_kernel3(int order, int nx, int ny, int nz,
    hipFloatComplex * dev_bandptr, hipFloatComplex * dev_bandptr2) {
//! Clear everything above and below zdistcutoff to 0

  int x1 = blockIdx.x * blockDim.x + threadIdx.x;
  if (x1 < nx/2+1) {
    int y1 = blockIdx.y;
    int z0 = blockIdx.z + const_zdistcutoff[order] + 1;

    int ind = z0*((nx/2+1)*ny) + y1 * (nx/2+1) + x1;
    dev_bandptr[ind] = make_hipFloatComplex(0.f, 0.f);
    if (order !=0)
      dev_bandptr2[ind] = make_hipFloatComplex(0.f, 0.f);
  }
  return;
}

__device__ hipFloatComplex dev_otfinterpolate(hipFloatComplex * otf, float kx, float ky, int kz, float kzscale)
  /* (kx, ky, kz) is Fourier space coords with origin at kx=ky=kz=0 and going  betwen -nx(or ny,nz)/2 and +nx(or ny,nz)/2 */
{
  hipFloatComplex otfval = make_hipFloatComplex(0.f, 0.f);
  // This should be rewritten using Textures for the interpolation. It will be much easier and faster!
  if (const_pParams_bRadAvgOTF) {
    int irindex, izindex, indices[2][2];
    float krindex, kzindex;
    float ar, az;

    krindex = sqrt(kx*kx+ky*ky) / const_pParams_dkrotf;
    kzindex = kz * kzscale;
    if (kzindex<0) kzindex += const_pParams_nzotf;

    irindex = floor(krindex);
    izindex = floor(kzindex);

    ar = krindex - irindex;
    az = kzindex - izindex;  // az is always 0 for 2D case, and it'll just become a 1D interp

    if (izindex == const_pParams_nzotf-1) {
      indices[0][0] = irindex*const_pParams_nzotf+izindex;
      indices[0][1] = irindex*const_pParams_nzotf;
      indices[1][0] = (irindex+1)*const_pParams_nzotf+izindex;
      indices[1][1] = (irindex+1)*const_pParams_nzotf;
    }
    else {
      indices[0][0] = irindex*const_pParams_nzotf+izindex;
      indices[0][1] = irindex*const_pParams_nzotf+(izindex+1);
      indices[1][0] = (irindex+1)*const_pParams_nzotf+izindex;
      indices[1][1] = (irindex+1)*const_pParams_nzotf+(izindex+1);
    }
    otfval.x = (1-ar)*(otf[indices[0][0]].x*(1-az) + otf[indices[0][1]].x*az) +
      ar*(otf[indices[1][0]].x*(1-az) + otf[indices[1][1]].x*az);
    otfval.y = (1-ar)*(otf[indices[0][0]].y*(1-az) + otf[indices[0][1]].y*az) +
      ar*(otf[indices[1][0]].y*(1-az) + otf[indices[1][1]].y*az);
  }
  return otfval;
}

__device__ float dev_order0damping(float radius, float zindex, float rlimit, int zlimit)
{
  float rfraction, zfraction;

  rfraction = radius/rlimit;
  zfraction = fabs(zindex/zlimit);

  return rfraction*rfraction + zfraction*zfraction*zfraction;
}

__device__ float dev_mag2(hipFloatComplex x)
{
  return x.x*x.x+x.y*x.y;
}

__device__ float dev_suppress(float x)
{
  float x6, out;
  x6 = x*x*x;
  x6 *= x6;
  out = 1.0/(1+20000/(x6+20));
  return out;
}

__host__ void assemblerealspacebands(int dir, GPUBuffer* outbuffer,
    GPUBuffer* bigbuffer, std::vector<GPUBuffer>* bands, int ndirs,
    int norders, const std::vector<vector>& k0, int nx, int ny, int nz,
    float dxy, float zoomfact, int z_zoom, float expfact)
{
  float fact,* dev_coslookup,* dev_sinlookup;
  int order;

  /* Allocate temporaries */    
  cutilSafeCall(hipMalloc((void **) &dev_coslookup,
                           (int)(rint(nx*zoomfact)*rint(ny*zoomfact)*sizeof(float))));
  cutilSafeCall(hipMalloc((void **) &dev_sinlookup,
                           (int)(rint(nx*zoomfact)*rint(ny*zoomfact)*sizeof(float))));

  fact = expfact/0.5;  // expfact is used for "exploded view".  For normal reconstruction expfact = 1.0

  int nThreads = 128;
  int NZblock = nz;
  int NYblock = ny;
  int NXblock = nx/nThreads;
  if (nx%nThreads) NXblock ++;

  dim3 grid(NXblock, NYblock, NZblock);
  dim3 block(nThreads, 1, 1);

  printf("moving centerband\n");
  cutilSafeCall(hipMemset((void*) bigbuffer->getPtr(), 0,
        unsigned (rint(zoomfact*nx)*rint(zoomfact*ny)*(z_zoom*nz)*sizeof(hipFloatComplex))));
  move_kernel<<<grid,block>>>(
      (hipFloatComplex*)bands->at(0).getPtr(),
      (hipFloatComplex*)bands->at(0).getPtr(),
      0, (hipFloatComplex*)bigbuffer->getPtr(), nx, ny, nz, zoomfact, z_zoom);


  // debug:
  size_t free, total;
  hipMemGetInfo(&free, &total);
  std::cout << "Before fftplan3d " << (free >> 20 ) << "MB free \n" ;
  //gubed

  hipfftHandle myGPUPlan;
  hipfftResult cuFFTErr = hipfftPlan3d(&myGPUPlan, (int) (z_zoom*nz), (int) rint(zoomfact*ny),
                                     (int) rint(zoomfact*nx), HIPFFT_C2C);
  if (cuFFTErr!=HIPFFT_SUCCESS) {
    if (cuFFTErr == HIPFFT_ALLOC_FAILED)
      printf("\n*** In assemblerealspacebands(), CUFFT failed to allocate GPU or CPU memory\n");
    throw std::runtime_error("CUFFT plan creation failed");
  }

  // debug:
  hipMemGetInfo(&free, &total);
  std::cout << "After fftplan " << (free >> 20 ) << "MB free \n" ;
  //gubed

  /* transform it */
  printf("re-transforming centerband\n");
  cuFFTErr = HIPFFT_SUCCESS;
  cuFFTErr = hipfftExecC2C(myGPUPlan,
      (hipFloatComplex*)bigbuffer->getPtr(),
      (hipFloatComplex*)bigbuffer->getPtr(),
      HIPFFT_BACKWARD);
  if (cuFFTErr!=HIPFFT_SUCCESS) printf("Error in hipfftExecC2C: %d\n", cuFFTErr);

  printf("inserting centerband\n");
  NZblock = (int)(z_zoom*nz);
  NYblock = (int) rint(zoomfact*ny);
  NXblock = (int) ceil((zoomfact*nx)/nThreads);
  dim3 grid2(NXblock, NYblock, NZblock);
  write_outbuffer_kernel1<<<grid2,block>>>((hipFloatComplex*)bigbuffer->getPtr(),
      (float*)outbuffer->getPtr(), (int) rint(zoomfact*nx));

  printf("centerband assembly completed\n");

  for (order=1; order < norders; order ++) {
    float k0x, k0y;
    /* move side bands to bigbuffer, fill in with zeroes */
    printf("moving order %d\n",order); 
    cutilSafeCall(hipMemset((void*) bigbuffer->getPtr(), 0, unsigned (rint(zoomfact*nx)*rint(zoomfact*ny)*(z_zoom*nz)*sizeof(hipFloatComplex))));
    move_kernel<<<grid,block>>>((hipFloatComplex*)bands->at(2*order-1).getPtr(),
        (hipFloatComplex*)bands->at(2*order).getPtr(), order,
        (hipFloatComplex*)bigbuffer->getPtr(), nx, ny, nz, zoomfact, z_zoom);

    // transform it into real space
    cuFFTErr = hipfftExecC2C(myGPUPlan, (hipFloatComplex*)bigbuffer->getPtr(),
        (hipFloatComplex*)bigbuffer->getPtr(), HIPFFT_BACKWARD);
    if (cuFFTErr!=HIPFFT_SUCCESS) printf("Error in hipfftExecC2C: %d\n", cuFFTErr);

    /***** For 3D, prepare 2D array of sines and cosines first, then loop over z. ******/
    k0x = k0[dir].x*((float)order);
    k0y = k0[dir].y*((float)order);

    NZblock = 1;
    NYblock = (int)rint (zoomfact*ny);
    NXblock = (int) ceil(zoomfact*nx/nThreads);
    dim3 grid3(NXblock, NYblock, NZblock);
    cos_sin_kernel<<<grid3,block>>>(k0x,  k0y, dxy/zoomfact, fact, dev_coslookup, dev_sinlookup, (int)(zoomfact*nx));
    // hipMemset((void*) dev_coslookup, 0, (unsigned)(rint(nx*zoomfact)*rint(ny*zoomfact)*sizeof(float)));
    // hipMemset((void*) dev_sinlookup, 0,  (unsigned)(rint(nx*zoomfact)*rint(ny*zoomfact)*sizeof(float)));
    write_outbuffer_kernel2<<<grid2, block>>>(dev_coslookup,
        dev_sinlookup, (hipFloatComplex*)bigbuffer->getPtr(), 
        (float*)outbuffer->getPtr(), (int) (zoomfact*nx));

    printf("order %d sideband assembly completed\n", order);
  } /* for (order =...) */

  /* Free memory */
  cutilSafeCall(hipFree((void *) dev_coslookup));
  cutilSafeCall(hipFree((void *) dev_sinlookup));
  hipfftDestroy(myGPUPlan);
  return;
}

__global__ void move_kernel(hipFloatComplex *inarray1, hipFloatComplex *inarray2, int order, 
    hipFloatComplex *outarray, int nx, int ny, int nz, float zoomfact, int z_zoom)
{
  int     xdim, ydim, zdim, nxy, nxyout;

  xdim=rint(zoomfact*nx); ydim=rint(zoomfact*ny);
  zdim = z_zoom*nz;
  nxy = (nx/2+1)*ny;
  nxyout = xdim*ydim;

  // compute x1, y1, z0 based on block and thread indices
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  if (x<nx) {
    x -= nx/2-1;
    int y = blockIdx.y - (ny/2-1);
    int z = (nz>1) ? blockIdx.z - (nz/2-1) : 0;

    int  indin, indout, xout, yout, zout, conj;
    hipFloatComplex valre, valim, val;

    xout = x;     /* xout,yot,zout = (non-centered) output coords with zoomed-up dims and origin of fourier space at (0,0,0) */
    if (xout<0) xout += xdim;
    yout = y;
    if (yout<0) yout += ydim;
    zout = z;
    if (zout<0) zout += zdim;
    indout = zout*nxyout + yout*xdim + xout;

    if (x<0) {    /* now xyz get turned into coords of the (half fourier space) input arrays */
      x = -x;
      y = -y;
      z = -z;
      conj = 1;
    }
    else
      conj = 0;

    if (y<0) y += ny;
    if (z<0) z += nz;
    indin = z*nxy + y*(nx/2+1) + x;

    if (order == 0) {
      val = inarray1[indin];
      if (conj)
        val.y *= -1;
    }
    else {
      valre = inarray1[indin];
      valim = inarray2[indin];
      if (conj) {
        valre.y *= -1;
        valim.y *= -1;
      }
      val.x = valre.x - valim.y;
      val.y = valre.y + valim.x;
    }

    outarray[indout] = val;
  }
  return;
}
__global__ void write_outbuffer_kernel1(hipFloatComplex * bigbuffer, float * outbuffer, int nx) {

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j<nx) {
    int i = blockIdx.y;
    int k = blockIdx.z;
    int NXlocal = nx;
    int NYlocal = gridDim.y;
    int ind = k*NXlocal*NYlocal + i*NXlocal + j;
    outbuffer[ind] += bigbuffer[ind].x;
  }
}

__global__ void write_outbuffer_kernel2(float * coslookup, float * sinlookup, 
      hipFloatComplex * bigbuffer, float * outbuffer, int nx) {

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j<nx) {
    int i = blockIdx.y;
    int k = blockIdx.z;
    int NXlocal = nx;
    int NYlocal = gridDim.y;
    int indxy = i*NXlocal + j;
    int ind = k*NXlocal*NYlocal + indxy;
    outbuffer[ind] += (bigbuffer[ind].x * 2.0*coslookup[indxy] - bigbuffer[ind].y * 2.0*sinlookup[indxy]);
  }
}

__global__ void cos_sin_kernel(float k0x, float k0y, float dxy, float fact, float * coslookup, float * sinlookup, int nx) {

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  int i = blockIdx.y;

  if (j<nx) {
    int ind = i*nx + j;
    int NXlocal = nx;
    int NYlocal = gridDim.y;
    float angle = fact * M_PI * ((j-NXlocal/2)*k0x + (i-NYlocal/2)*k0y) * dxy;
	sincosf(angle, sinlookup+ind, coslookup+ind);
  }
}

__host__ void computeAminAmax(const GPUBuffer* data, int nx, int ny, int nz,
    float* min, float* max)
{
  int numElems = nx * ny * nz;

  int blockSize = 1024;
  int numBlocks = 100;
  GPUBuffer maxPartialResult(numBlocks * sizeof(float), 0);
  GPUBuffer minPartialResult(numBlocks * sizeof(float), 0);
  computeAminAmax_kernel<<<numBlocks, blockSize,
    2 * blockSize * sizeof(float)>>>((const float*)data->getPtr(),
        numElems,
        (float*)maxPartialResult.getPtr(),
        (float*)minPartialResult.getPtr());
  CPUBuffer maxPartialResultHost(maxPartialResult.getSize());
  CPUBuffer minPartialResultHost(minPartialResult.getSize());
  maxPartialResult.set(&maxPartialResultHost, 0,
      maxPartialResult.getSize(), 0);
  minPartialResult.set(&minPartialResultHost, 0,
      minPartialResult.getSize(), 0);
  const float* maxArray = (const float*)maxPartialResultHost.getPtr();
  const float* minArray = (const float*)minPartialResultHost.getPtr();
  *max = -10000.0;
  *min = 10000.0;
  for (int i = 0; i < numBlocks; ++i) {
    if (maxArray[i] > *max) {
      *max = maxArray[i];
    }
    if (minArray[i] < *min) {
      *min = minArray[i];
    }
  }
}

__global__ void computeAminAmax_kernel(const float* data, int numElems,
    float* maxPartialResult, float* minPartialResult)
{
  volatile extern __shared__ float s_maxmin[];
  s_maxmin[threadIdx.x] = -10000.0f;
  s_maxmin[threadIdx.x + blockDim.x] = 10000.0f;
  for (int i = threadIdx.x;
      i < (int)ceil((float)numElems / (blockDim.x * gridDim.x));
      i += (blockDim.x * gridDim.x)) {
    if (i < numElems) {
      float d = data[i];
      if (d > s_maxmin[threadIdx.x]) {
        s_maxmin[threadIdx.x] = d;
      }
      if (d < s_maxmin[threadIdx.x + blockDim.x]) {
        s_maxmin[threadIdx.x + blockDim.x] = d;
      }
    }
  }
  __syncthreads();
  for (int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (threadIdx.x < s) {
      if (s_maxmin[threadIdx.x + s] > s_maxmin[threadIdx.x]) {
        s_maxmin[threadIdx.x] = s_maxmin[threadIdx.x + s];
      }
      if (s_maxmin[blockDim.x + threadIdx.x + s] < s_maxmin[blockDim.x + threadIdx.x]) {
        s_maxmin[blockDim.x + threadIdx.x] = s_maxmin[blockDim.x + threadIdx.x + s];
      }
    }
    __syncthreads();
  }
  if (threadIdx.x == 0) {
    maxPartialResult[blockIdx.x] = s_maxmin[0];
    minPartialResult[blockIdx.x] = s_maxmin[blockDim.x];
  }
}


// compute the mean above the background using GPU reduction
__host__ double meanAboveBackground_GPU(GPUBuffer &img, int nx, int ny, int nz)
{
  unsigned nThreads = 1024;
  unsigned nBlocks = (unsigned) ceil( nx*ny*nz /(float) nThreads/2 );
  unsigned smemSize = nThreads * sizeof(double);

  // used for holding intermediate reduction results; one for each thread block
  GPUBuffer d_intres(nBlocks * sizeof(double), 0);

  summation_kernel<<<nBlocks, nThreads, smemSize>>>((float *) img.getPtr(),
                                                    (double *) d_intres.getPtr(), nx*ny*nz);
  // download intermediate results to host:
  CPUBuffer intRes(d_intres);
  double sum=0;
  double *p=(double *)intRes.getPtr();
  for (int i=0; i<nBlocks; i++)
    sum += *p++;

  float mean = sum/((nx-2)*ny*nz);

  GPUBuffer d_counter(nBlocks * sizeof(unsigned), 0);
  smemSize = nThreads * (sizeof(double) + sizeof(unsigned));
  sumAboveThresh_kernel<<<nBlocks, nThreads, smemSize>>>((float *) img.getPtr(),
                                                         (double *) d_intres.getPtr(),
                                                         (unsigned *) d_counter.getPtr(),
                                                         mean, nx*ny*nz);
  
  // download intermediate results to host:
  CPUBuffer counter(d_counter);
  intRes = d_intres;
  sum=0;
  unsigned count = 0;
  p=(double *)intRes.getPtr();
  unsigned *pc = (unsigned *) counter.getPtr();
  for (int i=0; i<nBlocks; i++) {
    sum += *p++;
    count += *pc++;
  }

  // printf("mean=%e, sum=%e, count=%d\n", mean, sum, count);
  return sum/count;
}

// Utility class used to avoid linker errors with extern
// unsized shared memory arrays with templated type
// (Copied from reduction_kernel.cu of CUDA samples)
template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

__global__ void summation_kernel(float * img, double * intRes, int n)
// Copied from CUDA "reduction" sample code reduce4()
{
  double *sdata = SharedMemory<double>();

  unsigned tid = threadIdx.x;
  unsigned ind = blockIdx.x * blockDim.x*2 + threadIdx.x;

  double mySum= (ind < n) ? img[ind] : 0;

  if (ind + blockDim.x < n)
    mySum += img[ind + blockDim.x];

  sdata[tid] = mySum;
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid < 32) {
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile double *smem = sdata;

    // Assuming blockSize is > 64:
    smem[tid] += smem[(tid + 32)];
    smem[tid] += smem[(tid + 16)];
    smem[tid] += smem[(tid +  8)];
    smem[tid] += smem[(tid +  4)];
    smem[tid] += smem[(tid +  2)];
    smem[tid] += smem[(tid +  1)];
  }
  // write result for this block to global mem
  if (tid == 0) intRes[blockIdx.x] = sdata[0];
}


__global__ void sumAboveThresh_kernel(float * img, double * intRes, unsigned * counter, float thresh, int n)
// Adapted from CUDA "reduction" sample code reduce4()
{
// Size of shared memory allocated is nThreads * (sizeof(double) + sizeof(unsigned))
// The first nThreads * sizeof(double) bytes are used for image intensity sum;
// the next nThreads * sizeof(unsigned) bytes are for counting pixels whose intensity is > thresh
  double *sdata = SharedMemory<double>();
  unsigned *count = (unsigned *) (sdata + blockDim.x);

  unsigned tid = threadIdx.x;
  unsigned ind = blockIdx.x * blockDim.x*2 + threadIdx.x;

  double mySum= 0;
  unsigned myCount = 0;
  if (ind < n && img[ind] > thresh) {
    mySum = img[ind]; 
    myCount ++;
  }

  unsigned ind2 = ind + blockDim.x;
  if (ind2 < n && img[ind2] > thresh) {
    mySum += img[ind2];
    myCount ++;
  }

  sdata[tid] = mySum;
  count[tid] = myCount;
  __syncthreads();

  // do reduction in shared mem
  for (unsigned int s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
      count[tid] += count[tid + s];
    }
    __syncthreads();
  }

  if (tid < 32) {
    volatile double *smem = sdata;
    volatile unsigned *cmem = count;

    smem[tid] += smem[(tid + 32)];
    smem[tid] += smem[(tid + 16)];
    smem[tid] += smem[(tid +  8)];
    smem[tid] += smem[(tid +  4)];
    smem[tid] += smem[(tid +  2)];
    smem[tid] += smem[(tid +  1)];
    cmem[tid] += cmem[(tid + 32)];
    cmem[tid] += cmem[(tid + 16)];
    cmem[tid] += cmem[(tid +  8)];
    cmem[tid] += cmem[(tid +  4)];
    cmem[tid] += cmem[(tid +  2)];
    cmem[tid] += cmem[(tid +  1)];
  }
  // write result for this block to global mem
  if (tid == 0) {
    intRes[blockIdx.x] = sdata[0];
    counter[blockIdx.x] = count[0];
  }
}

__host__ void rescale_GPU(GPUBuffer &img, int nx, int ny, int nz, float scale)
{
  unsigned nThreads = 1024;
  unsigned nBlocks = (unsigned) ceil( nx*ny*nz / (float) nThreads );
  scale_kernel<<<nBlocks, nThreads>>>((float *) img.getPtr(), scale, nx*ny*nz);
#ifndef NDEBUG
  std::cout<< "rescale_GPU(): " << hipGetErrorString(hipGetLastError()) << std::endl;
#endif
}

__global__ void scale_kernel(float * img, double factor, int n)
{
  unsigned ind = blockIdx.x * blockDim.x + threadIdx.x;
  if (ind < n)
    img[ind] *= factor;
}

#include "hip/hip_runtime.h"
#ifndef GPU_FUNCTIONS_IMPL_H
#define GPU_FUNCTIONS_IMPL_H

#include "gpuFunctions.h"
#include <cmath>
#include <iostream>
#include <fstream>
#include <algorithm>
#include "../cutilSafeCall.h"

#ifndef RED_BLOCK_SIZE_X
#define RED_BLOCK_SIZE_X 32
#endif
#ifndef RED_BLOCK_SIZE_Y
#define RED_BLOCK_SIZE_Y 32
#endif


/** Data from pParams going into constant memory */
__constant__ int const_pParams_bSuppress_singularities;
__constant__ int const_pParams_suppression_radius;
__constant__ int const_pParams_bDampenOrder0;
__constant__ int const_pParams_bNoKz0;
__constant__ int const_pParams_bFilteroverlaps;
__constant__ int const_pParams_apodizeoutput;
__constant__ float const_pParams_apoGamma;
__constant__ bool const_pParams_bBessel;
__constant__ int const_pParams_bRadAvgOTF;
__constant__ int const_pParams_nzotf;
__constant__ float const_pParams_dkrotf;
__constant__ float const_wiener;

/** These data are not modified in the kernels and can go in constant memory */
__constant__ int const_zdistcutoff[3];
__constant__ float const_ampmag2[3];
__constant__ float const_ampmag2_alldirs[9];
__constant__ hipFloatComplex const_conjamp[3];
__constant__ float const_noiseVarFactors[9];
__constant__ float2 const_k0[3];
__constant__ hipFloatComplex * const_otfPtrs[3];

#define MAX_ORDERS 32
#define MAX_PHASES 32
__constant__ float* const_outputPtrs[MAX_ORDERS * 2 -1];
__constant__ float* const_imgPtrs[MAX_PHASES];
__constant__ float const_sepMatrix[(MAX_ORDERS * 2 - 1) * MAX_PHASES];

__global__ void image_arithmetic_kernel(float* a, const float* b,
    int len, float alpha, float beta);
__global__ void apodize_x_kernel(int napodize, int nx, int ny,
    float* image);
__global__ void apodize_y_kernel(int napodize, int nx, int ny,
    float* image);
__global__ void cosapodize_kernel(int nx, int ny, float* image, float cosPeriodX, float cosPeriodY, float cosAmpX, float cosAmpY);
__global__ void rescale_kernel(float* img, int nx, int ny,
    float scaleFactor);
__global__ void sum_reduction_kernel(float* img, int nx, int ny,
    float* partialReduction);

__host__ void makeoverlaps(std::vector<GPUBuffer>* bands,
    GPUBuffer* overlap0, GPUBuffer* overlap1, int nx, int ny, int nz,
    int order1, int order2,
    float k0x, float k0y, float dy, float dz,
    std::vector<GPUBuffer>* OTF, short wave, ReconParams* pParams);

__global__ void makeOverlaps0Kernel(int nx, int ny, int nz,
    int order1, int order2, float kx, float ky,
    float rdistcutoff, float otfcutoff, float zdistcutoff,
	float order0_2_factor, float dkx, float dky, float kzscale,
    hipFloatComplex *band1im, hipFloatComplex *band1re,
    hipFloatComplex *overlap0);
__global__ void makeOverlaps1Kernel(int nx, int ny, int nz,
    int order1, int order2, float kx, float ky,
    float rdistcutoff, float otfcutoff, float zdistcutoff,
	float order0_2_factor, float dkx, float dky, float kzscale,
    hipFloatComplex *band2im, hipFloatComplex *band2re,
    hipFloatComplex *overlap1);

__host__ void aTimesConjB(GPUBuffer* overlap0, GPUBuffer* overlap1,
    int nx, int ny, int nz, GPUBuffer* crosscorr_c);
__global__ void aTimesConjBKernel(hipFloatComplex* overlap0,
    hipFloatComplex* overlap1, int nx, int ny, int nz,
    hipFloatComplex* crosscorr_c);

__host__ void computeIntensities(GPUBuffer* amplitudes, int nx, int ny,
    GPUBuffer* intensities);
__global__ void computeIntensitiesKernel(hipFloatComplex* amplitudes,
    int nx, int ny, float* intensities);

__host__ void findpeak(float array[], int sizex, int sizey, vector *peak);
__host__ float fitparabola( float a1, float a2, float a3);

__host__ void computeIntensities(GPUBuffer* amplitudes, int nx, int ny,
    GPUBuffer* intensities);

__host__ float getmodamp(float kangle, float klength,
    std::vector<GPUBuffer>* bands, GPUBuffer* overlap0, GPUBuffer* overlap1,
    int nx, int ny,int nz, int order1, int order2, float dy, float dz,
    std::vector<GPUBuffer>* otf, short wave, hipFloatComplex* modamp,
    int redoarrays, ReconParams *pParams, int bShowDetail); 

__host__ float findrealspacemodamp(std::vector<GPUBuffer>* bands,
    GPUBuffer* overlap0, GPUBuffer* overlap1, int nx, int ny, int nz,
    int order1, int order2, vector k0, float dy, float dz,
    std::vector<GPUBuffer>* OTF, short wave, hipFloatComplex* modamp1,
    hipFloatComplex* modamp2, hipFloatComplex* modamp3, int redoarrays,
    ReconParams *pParams);

__global__ void reductionKernel(
    int nx, int ny, int nz,
    float kx, float ky, float dxy,
    const hipFloatComplex *overlap0, const hipFloatComplex *overlap1,
    hipFloatComplex *XStarY, float *sumXMag, float *sumYMag);

__host__ float fitxyparabola( float x1, float y1, float x2, float y2,
    float x3, float y3);

__device__ hipFloatComplex dev_otfinterpolate(hipFloatComplex * otf, float
    kx, float ky, int kz, float kzscale);

__device__ float dev_suppress(float x);
__device__ float dev_mag2(hipFloatComplex x);
__device__ float dev_order0damping(float radius, float zindex, float rlimit, int zlimit);
__global__ void move_kernel(hipFloatComplex *inarray1, hipFloatComplex *inarray2, int order, 
			    hipFloatComplex *outarray, int nx, int ny, int nz, float
          zoomfact, int z_zoom);
__global__ void write_outbuffer_kernel1(hipFloatComplex * bigbuffer,
                                        float * outbuffer, int);
__global__ void write_outbuffer_kernel2(float * coslookup, float * sinlookup, 
                                        hipFloatComplex * bigbuffer, float * outbuffer, int);

__global__ void cos_sin_kernel(float k0x, float k0y, float dxy, float fact,
							   float * coslookup, float * sinlookup, int);

__global__ void filterbands_kernel1(int dir, int ndirs, int order, int norders, int nx, 
    int ny, int nz, float rdistcutoff, float zapocutoff, float apocutoff, 
	float krscale, float kzscale,
    hipFloatComplex * dev_bandptr, hipFloatComplex * dev_bandptr2, bool bSecondEntry);

// __global__ void filterbands_kernel2(int dir, int ndirs, int order, int norders, int nx, 
//     int ny, int nz, float rdistcutoff, float zapocutoff, float apocutoff, 
// 	float krscale, float kzscale, /*hipFloatComplex *dev_scale,*/ hipFloatComplex * dev_tempbandplus,
//     hipFloatComplex * dev_bandptr, hipFloatComplex * dev_bandptr2);

__global__ void filterbands_kernel3(int order, int nx, int ny, int nz,
				    hipFloatComplex * dev_bandptr, hipFloatComplex *
            dev_bandptr2);

__global__ void filterbands_kernel4(int order, int nx, int ny, int nz,
    hipFloatComplex * dev_tempbandplus, hipFloatComplex * dev_bandptr,
    hipFloatComplex * dev_bandptr2);

__global__ void separate_kernel(int norders, int nphases, int nx, int ny, int nz);
__global__ void computeAminAmax_kernel(const float* data, int numElems,
    float* maxPartialResult, float* minPartialResult);

__global__ void summation_kernel(float * img, double * intRes, int n);
__global__ void sumAboveThresh_kernel(float * img, double * intRes, unsigned * counter, float thresh, int n);
__global__ void scale_kernel(float * img, double factor, int n);


template<typename T>
T cpuReduce(const T* vec, int n) {
  T red = 0;
  for (int i = 0; i < n; ++i) {
    red += vec[i];
  }
  return red;
}

hipFloatComplex cpuReduce(const hipFloatComplex* vec, int n) {
  hipFloatComplex red;
  red.x = 0;
  red.y = 0;
  for (int i = 0; i < n; ++i) {
    red.x += vec[i].x;
    red.y += vec[i].y;
  }
  return red;
}
#endif
